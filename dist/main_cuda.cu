#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


__constant__ double c_center_y, c_center_x;
__constant__ int c_width, c_height;

__global__ void mandelbrot_kernel(uint8_t *output, double x_start, double x_finish, double y_start, double y_finish)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= c_height || col >= c_width)
    {
        return;
    }
    int max_iter = 256;
    int iter = 0;

    double dx = (x_finish - x_start) / c_width;
    double dy = (y_finish - y_start) / c_height;
    int idx = row * c_width + col;
    double real = x_start + col * dx;
    double imag = y_start + row * dy;
    double c_real = real;
    double c_imag = imag;

    double real2, imag2;

    for (int i = 0; i < max_iter; ++i)
    {
        real2 = real * real;
        imag2 = imag * imag;
        if (real2 + imag2 > 4.0)
        {
            break;
        }
        imag = 2 * real * imag + c_imag;
        real = real2 - imag2 + c_real;
        iter++;
    }

    double t = (double)iter / max_iter;
    uint8_t r, g, b;

    if (iter == max_iter)
    {
        r = g = b = 0; // 黑色
    }
    else
    {
        double t1 = 1 - t;
        r = static_cast<uint8_t>(9 * t1 * t * t * t * 255);
        g = static_cast<uint8_t>(15 * t1 * t1 * t * t * 255);
        b = static_cast<uint8_t>(8.5 * t1 * t1 * t1 * t * 255);
    }

    output[idx * 3] = r;
    output[idx * 3 + 1] = g;
    output[idx * 3 + 2] = b;
}

class MandelbrotCUDA
{
public:
    MandelbrotCUDA(int width, int height) : width(width), height(height)
    {
        ratio = static_cast<double>(width) / height;
        initCUDA();
    }

#define TILE_WIDTH 32
    void compute(double x_start, double x_finish, double y_start, double y_finish)
    {
        dim3 dimGrid(ceil((double)width / TILE_WIDTH), ceil((double)height / TILE_WIDTH), 1);
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
        uint8_t *dataptr = thrust::raw_pointer_cast(&data_device[0]);

        mandelbrot_kernel<<<dimGrid, dimBlock>>>(dataptr, x_start, x_finish, y_start, y_finish);

        data_host = data_device;
    }

    uint8_t *get_data()
    {
        return data_host.data();
    }

    void set_zoom(double zoom)
    {
        this->zoom = zoom;
    }
    void set_center(double center_x, double center_y)
    {
        this->center_x = center_x;
        this->center_y = center_y;
    }
    void set_boundaries(double x_start, double x_finish, double y_start, double y_finish)
    {
        this->x_start = x_start;
        this->x_finish = x_finish;
        this->y_start = y_start;
        this->y_finish = y_finish;
    }

    void update()
    {
        this->scale *= zoom;
        this->x_start = center_x - 0.5 * ratio * scale;
        this->x_finish = center_x + 0.5 * ratio * scale;
        this->y_start = center_y - 0.5 * scale;
        this->y_finish = center_y + 0.5 * scale;
    }

    void initCUDA()
    {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0)
        {
            std::cerr << "No CUDA devices found." << std::endl;
            exit(1);
        }
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        std::cout << "Using CUDA device: " << deviceProp.name << std::endl;

        data_host.resize(width * height * 3);
        data_device = data_host;

        hipMemcpyToSymbol(HIP_SYMBOL(c_center_x), &this->center_x, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(c_center_y), &this->center_y, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(c_width), &this->width, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(c_height), &this->height, sizeof(int));
    }

private:
    int width, height;
    thrust::host_vector<uint8_t> data_host;
    thrust::device_vector<uint8_t> data_device;
    double center_x, center_y;
    double scale = 1.0;
    double ratio;
    double zoom;
    double x_start, x_finish, y_start, y_finish;
};
